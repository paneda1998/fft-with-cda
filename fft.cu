#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"
#include "math.h"
#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z
#define Pi 3.141592654f
#define pi 3.141592654f

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

//-----------------------------------------------------------------------------


__device__ unsigned int reverse4(register unsigned int x,const unsigned int M)
	
{
   // x = (((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1));
    x = (((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2));
    x = (((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4));
    x = (((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8));
    x = ((x >> 16) | (x << 16));
    x>>=(32-M);
return x;


}
__device__ unsigned int reverse2(register unsigned int x,const unsigned int M)
	
{
    x = (((x & 0xaaaaaaaa) >> 1) | ((x & 0x55555555) << 1));
    x = (((x & 0xcccccccc) >> 2) | ((x & 0x33333333) << 2));
    x = (((x & 0xf0f0f0f0) >> 4) | ((x & 0x0f0f0f0f) << 4));
    x = (((x & 0xff00ff00) >> 8) | ((x & 0x00ff00ff) << 8));
    x = ((x >> 16) | (x << 16));
    x>>=(32-M);
return x;


}

__global__ void reverseb2(float* input,float* input1,const unsigned int N, unsigned int M) //radix-2
{

	float temp;
	float temp1;
    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;
  
unsigned int reverse_num=reverse2(i,M);
    if(i<reverse_num)
{
temp=input[reverse_num];
temp1=input1[reverse_num];

input[reverse_num]=input[i];
input1[reverse_num]=input1[i];

input[i]=temp;
input1[i]=temp1;
}

}
__global__ void reverseb4(float* input,float* input1,const unsigned int N, unsigned int M) //radix-2
{

	float temp;
	float temp1;
    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;
      //int i =  bx * 1024  + tx;
unsigned int reverse_num=reverse4(i,M);

    if(i<reverse_num)
{
temp=input[reverse_num];
temp1=input1[reverse_num];

input[reverse_num]=input[i];
input1[reverse_num]=input1[i];

input[i]=temp;
input1[i]=temp1;
}

}



__global__ void kernelFunc(float* x_r_d, float* x_i_d ,const unsigned int N, unsigned int M) //radix-2
{
//int i = (by * gridDim.x + bx) * blockDim.x  + tx;
    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;
	int S = M;
	float WnR;                   
	float WnI;
    //int i =  bx * 1024  + tx;
	float temp_r_1, temp_r_2, temp_i_1, temp_i_2;
	
	temp_r_1 = x_r_d[i+(i/S)*S];         // in each step every butterfly take two inputs 
	temp_r_2 = x_r_d[i+(i/S)*S+(S)];
	
	WnR = cos(-1 * (2 * pi) * ((i*(N/(2*S)))-(i/S)*(N/2)) / (N));      //  wn = e ^ (-j * 2 * pi*  k / N)
	WnI = sin(-1 * (2 * pi) * ((i*(N/(2*S)))-(i/S)*(N/2)) / (N));	
	
	temp_i_1 = x_i_d[i+(i/S)*S];
	temp_i_2 = x_i_d[i+(i/S)*S+(S)];
   
	x_r_d[i+(i/S)*S] = temp_r_1 + WnR * temp_r_2 - WnI * temp_i_2;
	x_i_d[i+(i/S)*S] = temp_i_1 + WnR * temp_i_2 + WnI * temp_r_2;
	
	x_r_d[i+(i/S)*S+(S)] = temp_r_1 - WnR * temp_r_2 + WnI * temp_i_2;
	x_i_d[i+(i/S)*S+(S)] = temp_i_1 - WnR * temp_i_2 - WnI * temp_r_2;		
	
	
}




__global__ void kernelFunc2(float* x_r_d, float* x_i_d, const unsigned int N, const unsigned int M) //radix-4
{
	//...
	int S = M;
    //int i = (by * gridDim.x + bx) * blockDim.x  + tx;
    int i = (bz*gridDim.y*gridDim.x + by * gridDim.x + bx) * blockDim.x  + tx;
	float temp_r_1, temp_r_2, temp_i_1, temp_i_2, temp_r_4, temp_r_3, temp_i_4, temp_i_3;	
	float v_i_1, v_i_2, v_i_3, v_i_4, v_r_1, v_r_2, v_r_3, v_r_4;
	float angle  = -2*pi*(i%S) / (S*4);	
	
	
	temp_r_1 = x_r_d[((i/S)*(4*S)+(i%S))];         // in each step every butterfly take two inputs 
	temp_r_2 = x_r_d[((i/S)*(4*S)+(i%S))+(S)];
	temp_r_3 = x_r_d[((i/S)*(4*S)+(i%S)) + 2 * (S)];
	temp_r_4 = x_r_d[((i/S)*(4*S)+(i%S)) + 3 * (S)];
	
	temp_i_1 = x_i_d[((i/S)*(4*S)+(i%S))];
	temp_i_2 = x_i_d[((i/S)*(4*S)+(i%S)) + (S)];
	temp_i_3 = x_i_d[((i/S)*(4*S)+(i%S)) + 2 * (S)];
	temp_i_4 = x_i_d[((i/S)*(4*S)+(i%S)) + 3 * (S)];	
	
	v_r_1 = temp_r_1;
	v_i_1 = temp_i_1;
	
	v_r_2 = temp_r_2 * cos(angle) - temp_i_2 * sin(angle);
	v_i_2 = temp_r_2 * sin(angle) + temp_i_2 * cos(angle);

	v_r_3 = temp_r_3 * cos(2*angle) - temp_i_3 * sin(2*angle);
	v_i_3 = temp_r_3 * sin(2*angle) + temp_i_3 * cos(2*angle);

	v_r_4 = temp_r_4 * cos(3*angle) - temp_i_4 * sin(3*angle);
	v_i_4 = temp_r_4 * sin(3*angle) + temp_i_4 * cos(3*angle);	
	
	//int index = (i/S)*S*4+i%S;
	//__syncthread;
	x_r_d[(i/S)*S*4+i%S] = v_r_1 + v_r_2 + v_r_3 + v_r_4;
	x_i_d[(i/S)*S*4+i%S] = v_i_1 + v_i_2 + v_i_3 + v_i_4;
	
	x_r_d[(i/S)*S*4+i%S + S] = v_r_1 + v_i_2 - v_r_3 - v_i_4;
	x_i_d[(i/S)*S*4+i%S + S] = v_i_1 - v_r_2 - v_i_3 + v_r_4;
	
	x_r_d[(i/S)*S*4+i%S + 2 * S] = v_r_1 - v_r_2 + v_r_3 - v_r_4;
	x_i_d[(i/S)*S*4+i%S + 2 * S] = v_i_1 - v_i_2 + v_i_3 - v_i_4;
	
	x_r_d[(i/S)*S*4+i%S + 3 * S] = v_r_1 - v_i_2 - v_r_3 + v_i_4;
	x_i_d[(i/S)*S*4+i%S + 3 * S] = v_i_1 + v_r_2 - v_i_3 - v_r_4;
	
}








void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
 
int S=0;

	

	dim3 dimGrid1((N / (512*512)), 32, 32);
	dim3 dimBlock1(1024/4, 1, 1);


	if((M==24) | (M==26))
{
	dim3 dimGrid3((N / (1024 *256*4)), 32, 32);

	dim3 dimBlock3(1024/4, 1, 1);
        reverseb4 <<<  dimGrid1, dimBlock1  >>>(x_r_d, x_i_d, N, M);
	for ( S = 1; S < N; S*=4)  //  stage = log S
	{
	
	    kernelFunc2 <<< dimGrid3, dimBlock3 >>>(x_r_d, x_i_d, N, S);

		
		}

}
else
{
	dim3 dimGrid((N / (1024 *256*2)), 32, 32);
	dim3 dimBlock(1024/4, 1, 1);
        reverseb2 <<<  dimGrid1, dimBlock1  >>>(x_r_d, x_i_d, N, M);
	for ( S = 1; S < N; S*=2)  //  stage = log S
	{
	
	    kernelFunc <<< dimGrid, dimBlock >>>(x_r_d, x_i_d, N, S);

		
		}



}

	
}
